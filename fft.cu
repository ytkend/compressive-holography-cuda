#include "fft.cuh"

namespace gpu {

FFT::FFT(int n) : n0_(n), n1_(1)
{
    hipfftPlan1d(&plan_, n, HIPFFT_C2C, 1);
}

FFT::FFT(int n0, int n1) : n0_(n0), n1_(n1)
{
    hipfftPlan2d(&plan_, n0, n1, HIPFFT_C2C);
}

FFT::~FFT()
{
    hipfftDestroy(plan_);
}

void FFT::forward(const hipfftComplex* in, hipfftComplex* out)
{
    hipfftExecC2C(plan_, const_cast<hipfftComplex*>(in), out, HIPFFT_FORWARD);
}

void FFT::forward(const thrust::complex<float>* in, thrust::complex<float>* out)
{
    forward(reinterpret_cast<const hipfftComplex*>(in), reinterpret_cast<hipfftComplex*>(out));
}

void FFT::inverse(const hipfftComplex* in, hipfftComplex* out)
{
    hipfftExecC2C(plan_, const_cast<hipfftComplex*>(in), out, HIPFFT_BACKWARD);
    float c = 1.0 / (n0_ * n1_);
    thrust::transform(thrust::device_ptr<hipfftComplex>(out),
                      thrust::device_ptr<hipfftComplex>(out) + n0_ * n1_,
                      thrust::device_ptr<hipfftComplex>(out),
                      [c] __host__ __device__ (hipfftComplex z) { return hipfftComplex{c * z.x, c * z.y}; });
}

void FFT::inverse(const thrust::complex<float>* in, thrust::complex<float>* out)
{
    inverse(reinterpret_cast<const hipfftComplex*>(in), reinterpret_cast<hipfftComplex*>(out));
}

FFTMany::FFTMany(int n, int batch) : n0_(n), n1_(1)
{
    hipfftPlan1d(&plan_, n, HIPFFT_C2C, batch);
}

FFTMany::FFTMany(int n0, int n1, int batch) : n0_(n0), n1_(n1), batch_(batch)
{
    int rank = 2;
    int n[] = {n0, n1};
    int idist = n0 * n1;
    int odist = n0 * n1;
    hipfftPlanMany(&plan_, rank, n,
                  NULL, 1, idist,
                  NULL, 1, odist,
                  HIPFFT_C2C, batch);
}

FFTMany::~FFTMany()
{
    hipfftDestroy(plan_);
}

void FFTMany::forward(const hipfftComplex* in, hipfftComplex* out)
{
    hipfftExecC2C(plan_, const_cast<hipfftComplex*>(in), out, HIPFFT_FORWARD);
}

void FFTMany::forward(const thrust::complex<float>* in, thrust::complex<float>* out)
{
    forward(reinterpret_cast<const hipfftComplex*>(in), reinterpret_cast<hipfftComplex*>(out));
}

void FFTMany::inverse(const hipfftComplex* in, hipfftComplex* out)
{
    hipfftExecC2C(plan_, const_cast<hipfftComplex*>(in), out, HIPFFT_BACKWARD);
    float c = 1.0 / (n0_ * n1_);
    thrust::transform(thrust::device_ptr<hipfftComplex>(out),
                      thrust::device_ptr<hipfftComplex>(out) + n0_ * n1_ * batch_,
                      thrust::device_ptr<hipfftComplex>(out),
                      [c] __host__ __device__ (hipfftComplex z) { return hipfftComplex{c * z.x, c * z.y}; });
}

void FFTMany::inverse(const thrust::complex<float>* in, thrust::complex<float>* out)
{
    inverse(reinterpret_cast<const hipfftComplex*>(in), reinterpret_cast<hipfftComplex*>(out));
}

}
